#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define THREADS_PER_BLOCK 32

__global__ void BackwardPass_Kernel(
    const float* __restrict__ logits, //nrows, vocab_size
    const int* __restrict__ labels, //nrows,
    const float* __restrict__ logsumexp, // nrows,
    float* __restrict__ dlogits, //nrows, vocab_size
    int nrows, 
    int vocab_size
){
    int row = blockIdx.x;
    int tid = threadIdx.x;

    const float* row_logits = logits + row*vocab_size;
    float* row_dlogits = dlogits + row*vocab_size;

    float logsumexpval = logsumexp[row];

    // Computing the softmax
    for(int i = tid; i < vocab_size; i += blockDim.x){
        row_dlogits[i] = expf(row_logits[i] - logsumexpval); // p_i = exp(x_i - logsumexp)
    }

    if(tid == 0){
        int label = labels[row];
        if(label >= 0){
            row_dlogits[label] -= 1.0f;
        }
    }
}

void compute_cross_entropy_backward(float *h_logits, int *h_labels, float *h_logsumexp, float *h_dlogits, int n_rows, int vocab_size){
    float *d_logits, *d_dlogits, *d_logsumexp;
    int *d_labels;

    hipMalloc(&d_logits, n_rows * vocab_size * sizeof(float));
    hipMalloc(&d_dlogits, n_rows * vocab_size * sizeof(float));
    hipMalloc(&d_labels, n_rows * sizeof(int));
    hipMalloc(&d_logsumexp, n_rows * sizeof(float));

    hipMemcpy(d_logits, h_logits, n_rows * vocab_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_labels, h_labels, n_rows * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_logsumexp, h_logsumexp, n_rows * sizeof(float), hipMemcpyHostToDevice);
    
    BackwardPass_Kernel<<<n_rows, THREADS_PER_BLOCK>>>(
        d_logits, d_labels, d_dlogits, d_logsumexp, n_rows, vocab_size);
    hipDeviceSynchronize();

    hipMemcpy(h_dlogits, d_dlogits, n_rows * vocab_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_logits);
    hipFree(d_labels);
    hipFree(d_logsumexp);
    hipFree(d_dlogits);
}

int main() {
    int n_rows = 2, vocab_size = 10; 
    float h_logits[] = {
        2.1, 1.0, 0.5, -0.5, 1.2, 0.8, 0.2, 0.3, -1.0, 0.7,
        1.8, 2.4, -0.2, 1.1, 0.9, 0.5, -0.3, 0.6, 0.0, 1.0
    };
    int h_labels[] = {2, 5}; // Example labels (correct class index)
    float h_logsumexp[] = {3.3026, 3.7883}; // Precomputed logsumexp
    float h_dlogits[20];

    compute_cross_entropy_backward(h_logits, h_labels, h_logsumexp, h_dlogits, n_rows, vocab_size);

    for (int i = 0; i < n_rows; i++) {
        printf("Gradients for row %d:\n", i);
        for (int j = 0; j < vocab_size; j++) {
            printf("%f ", h_dlogits[i * vocab_size + j]);
        }
        printf("\n");
    }

    return 0;
}