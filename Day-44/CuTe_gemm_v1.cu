#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


template <class ProblemShape, // {M, N, K}
        class CtaTiler, // Shape or layout to split data into tiles (blocks)
        class TA, class AStride, class ASmemLayout, class AThreadLayout, // Type, Stride, Shared Memory and Thread block layout
        class TB, class BStride, class BSmemLayout, class BThreadLayout,
        class TC, class CStride, class CSmemLayout, class CThreadLayout,
        class Alpha, class Beta> // Scalars used in epilogue

__global__ static 
__launch_bounds__(decltype(size(CThreadLayout{}))::value) // Informs the compiler: kernel always launches with exactly size(CThreadLayout) threads
void
gemm_device(ProblemShape shape_MNK,
            CtaTiler cta_tiler,
            TA const *A, AStride dA, ASmemLayout sA_layout, AThreadLayout tA,
            TB const *B, BStride dB, BSmemLayout sB_layout, BThreadLayout tB,
            TC * C, CStride dC, CSmemLayout sC_layout, CThreadLayout tC,
            Alpha alpha, Beta beta)
{
    using namespace cute;
    CUTE_STATIC_ASSERT_V(rank(shape_MNK) == INT<3>{}); / {M,N,K}

    CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA)); // {M, K}
    CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB)); //{N,K} and not {K,N} as per BLAS convention
    CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC)); // {M, N}

    // Representing the full tensors
    Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // {M,K} with stride dA = {1, lda} for nt {lda,1} for tn
    Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // {N,K} with stride dB = {1, ldb} for nt {ldb,1} for tn
    Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // {M,N} with stride = {1,ldc} for nt and {ldc,1} for tn

    // Getting the blocks for this thread block
    auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _); // (m, n, k)
    // k not specified since it will be looped over, we'll be needing all tiles
    
    // Global tensor reference, tile shape, which row and column specifically within the block, which modes to move along
    Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X, _1>{}); // {BLK_M, BLK_K}, In step we are specifying that we will be moving along M and K modes
    // This means local_tile(mA, select<0,2>(cta_tiler), select<0,2>(cta_coord))
    Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step<X, _1, _1>{}); // {BLK_K, BLK_N}
    Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1, _1, X>{}); // {BLK_M, BLK_N}

    // Shared memory buffers
    __shared__ TA smemA[cosize_v<ASmemLayout>]; // cosize will give total elements in ASmemLayout. 
    __shared__ TB smemB[cosize_v<BSmemLayout>];

    Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout); // BLOCK_SIZE_M , BLOCK_SIZE_K
    Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout); // BLOCK_SIZE_N , BLOCK_SIZE_K

    
}


// M-major smem sA, n-major smem sB, mn-major threads tA | tB
// M and N are column major
// As per convention in the docs, a matrix is X-major if stride in X-mode is 1
template <class TA, class TB, class TC, class Alpha, class Beta>
void gemma_nt(int m, int n, int k,
            Alpha alpha,
            TA const* A, int ldA,
            TB const* B, int ldB,
            Beta beta,
            TC *C, int ldC,
            hipStream_t stream = 0)
{
    using namespace cute;

    auto M = int(m);
    auto N = int(n);
    auto K = int(k);

    auto prob_shape = make_shape(M,N,K);

    // Defining strides (column major)
    auto dA = make_stride(Int<1>{}, ldA); // {dM, dK} {1,ldA}
    auto dB = make_stride(Int<1>{}, ldB); // {dN, dK} {1, ldB}
    auto dC = make_stride(Int<1>{}, ldC); // {dM, dN} {1, ldC}

    // Defining Tile Sizes
    auto bM = Int<128>{};
    auto bN = Int<128>{};
    auto bK = Int<8>{};

    auto cta_tiler = make_shape(bM, bN, bK); // {BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K}
    // This will be fed to a composition function to obtain tiles

    // Defining the shared memory layouts
    // Static layouts arse more efficient and allow CuTe to have optimized implementations

    auto sA = make_layout(make_shape(bM, bK)); // {BLOCK_SIZE_M, BLOCK_SIZE_K}, M-major
    auto sB = make_layout(make_shape(bN, bK)); // {BLOCK_SIZE_N, BLOCK_SIZE_K}, N-major
    auto sC = make_layout(make_shape(bM, bN)); // {BLOCK_SIZE_M, BLOCK_SIZE_N}, M-major

    // Defining thread layouts
    auto tA = make_layout(make_shape(Int<32>{}, Int<8>{}));
    auto tB = make_layout(make_shape(Int<32>{}, Int<8>{}));
    auto tC = make_layout(make_shape(Int<16>{}, Int<16>{}));

    dim3 dimBlock(size(tC)); // This matches the __launch__bounds before the kernel 

    dim3 dimGrid(size(ceil_div(M, bM)), 
                size(ceil_div(N, bN)));
    
    gemm_device<<<dimGrid, dimBlock 0, stream>>>(
        prob_shape, cta_tiler,
        A, dA, sA, tA,
        B, dB, sB, tB,
        C, dC, sC, tC,
        alpha, beta);
}
