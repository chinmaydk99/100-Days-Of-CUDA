#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

__global__ void sigmoid_kernel(float *z, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        z[i] = 1.0f / (1.0f + expf(-z[i]));
    }
}

void logistic_regression_cublas(float *d_X, float *d_y, float *d_w, 
                                int n, int d, float learning_rate, int epochs) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;
    float neg_learning_rate = -learning_rate;

    float *d_pred, *d_grad;
    hipMalloc(&d_pred, n * sizeof(float));  // Stores predictions
    hipMalloc(&d_grad, d * sizeof(float));  // Stores gradient

    for (int epoch = 0; epoch < epochs; epoch++) {
        hipblasSgemv(handle, HIPBLAS_OP_T, n, d, &alpha, d_X, d, d_w, 1, &beta, d_pred, 1);

        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;
        sigmoid_kernel<<<numBlocks, blockSize>>>(d_pred, n);

        float minus_one = -1.0f;
        hipblasSaxpy(handle, n, &minus_one, d_y, 1, d_pred, 1);

        hipblasSgemv(handle, HIPBLAS_OP_N, d, n, &alpha, d_X, n, d_pred, 1, &beta, d_grad, 1);

        hipblasSaxpy(handle, d, &neg_learning_rate, d_grad, 1, d_w, 1);

        if (epoch % 100 == 0) {
            float loss = 0.0f;
            hipblasSnrm2(handle, n, d_pred, 1, &loss);
            cout << "Epoch: " << epoch << " Loss: " << (loss / n) << endl;
        }
    }

    hipFree(d_pred);
    hipFree(d_grad);
    hipblasDestroy(handle);
}


int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const int n = 3;
    const int d = 2;
    const float learning_rate = 0.1f;
    const int epochs = 1000;

    float h_X[] = {1.0f, 2.0f,
                   3.0f, 4.0f,
                   5.0f, 6.0f};

    float h_y[] = {0.0f, 1.0f, 1.0f};


    float h_w[d] = {0.0f, 0.0f};

    float *d_X, *d_y, *d_w;
    hipMalloc(&d_X, n * d * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_w, d * sizeof(float));

    hipMemcpy(d_X, h_X, n * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_w, 0, d * sizeof(float));  // Initialize weights to zero

    logistic_regression_cublas(d_X, d_y, d_w, n, d, learning_rate, epochs);

    hipMemcpy(h_w, d_w, d * sizeof(float), hipMemcpyDeviceToHost);

    cout << "Final trained weights: ";
    for (int i = 0; i < d; i++) {
        cout << h_w[i] << " ";
    }
    cout << endl;


    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_w);
    hipblasDestroy(handle);

    return 0;
}
