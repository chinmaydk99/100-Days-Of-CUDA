#include "hip/hip_runtime.h"
// More work per thread. Each thread handles multiple outputs

#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cstring>
#include <cmath> 

#define BLOCK_SIZE 32

#define CUDA_CHECK(call) do{   \
    hipError_t e = call;      \
    if(e != hipSuccess){       \
        std::cerr << "CUDA error " << e << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while(0)

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
    sgemm2DBlocktiling(int M, int N, int K, float alpha, const float *A,
                       const float *B, float beta, float *C) {
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;

  const uint totalResultsBlocktile = BM * BN;
  // A thread is responsible for calculating TM*TN elements in the blocktile
  const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN);

  // ResultsPerBlock / ResultsPerThread == ThreadsPerBlock
  assert(numThreadsBlocktile == blockDim.x);

  // BN/TN are the number of threads to span a column
  const int threadCol = threadIdx.x % (BN / TN);
  const int threadRow = threadIdx.x / (BN / TN);

  // allocate space for the current blocktile in smem
  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];

  // Move blocktile to beginning of A's row and B's column
  A += cRow * BM * K;
  B += cCol * BN;
  C += cRow * BM * N + cCol * BN;

  // calculating the indices that this thread will load into SMEM
  const uint innerRowA = threadIdx.x / BK;
  const uint innerColA = threadIdx.x % BK;
  // calculates the number of rows of As that are being loaded in a single step
  // by a single block
  const uint strideA = numThreadsBlocktile / BK;
  const uint innerRowB = threadIdx.x / BN;
  const uint innerColB = threadIdx.x % BN;
  // for both As and Bs we want each load to span the full column-width, for
  // better GMEM coalescing (as opposed to spanning full row-width and iterating
  // across columns)
  const uint strideB = numThreadsBlocktile / BN;

  // allocate thread-local cache for results in registerfile
  float threadResults[TM * TN] = {0.0};
  // register caches for As and Bs
  float regM[TM] = {0.0};
  float regN[TN] = {0.0};

  // outer-most loop over block tiles
  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    // populate the SMEM caches
    for (uint loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
      As[(innerRowA + loadOffset) * BK + innerColA] =
          A[(innerRowA + loadOffset) * K + innerColA];
    }
    for (uint loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
      Bs[(innerRowB + loadOffset) * BN + innerColB] =
          B[(innerRowB + loadOffset) * N + innerColB];
    }
    __syncthreads();

    // advance blocktile
    A += BK;     // move BK columns to right
    B += BK * N; // move BK rows down

    // calculate per-thread results
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
      // block into registers
      for (uint i = 0; i < TM; ++i) {
        regM[i] = As[(threadRow * TM + i) * BK + dotIdx];
      }
      for (uint i = 0; i < TN; ++i) {
        regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
      }
      for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
          threadResults[resIdxM * TN + resIdxN] +=
              regM[resIdxM] * regN[resIdxN];
        }
      }
    }
    __syncthreads();
  }

  // write out the results
  for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
      C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN] =
          alpha * threadResults[resIdxM * TN + resIdxN] +
          beta * C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN];
    }
  }
}


void initialise_matrix(float * mat, int rows, int cols){
    std::random_device rd;
    std::mt19937 gen(rd()); // Mersenne Twister engine for generating pseudo random numbers
    std::uniform_real_distribution<> distrib(0.0f, 1.0f);

    for(int i = 0; i < rows * cols; ++i){
        mat[i] = static_cast<float>(distrib(gen));
    }
}

// Use const for input arrays
void gemm_cpu(const float *A, const float *B, float * C, int M, int N, int K, float alpha, float beta){
    for(int i = 0; i < M; i ++){
        for(int j = 0; j < N; j ++){
            float sum =  0.0f;
            for(int l = 0; l < K; l ++){
                sum += A[i * K + l] * B[l * N + j];
            }
            float initial_C = (beta == 0.0f) ? 0.0f : C[i * N + j];
            C[i * N + j] = alpha * sum + beta * initial_C;
        }
    }
}


int main(){
    int M = 1024;
    int N = 1024;
    int K = 512;
    float alpha = 1.0f;
    float beta = 1.0f;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    // Block parameters
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 2;

    float *h_A = new float[M*K];
    float *h_B = new float[K*N];
    float *h_C = new float[M*N];
    float *h_C_cpu = new float[M*N]; // Allocate memory for CPU result

    if(!h_A || !h_B || !h_C || !h_C_cpu){ // Updated check
        std::cerr << "Memory allocation failed" <<std::endl;
        delete[] h_A;
        delete[] h_B;
        delete[] h_C;
        delete[] h_C_cpu; // Added cleanup
        return -1;
    }
    std::cout<<"Host side memory allocated" << std::endl;

    initialise_matrix(h_A, M, K);
    initialise_matrix(h_B, K, N);
    initialise_matrix(h_C, M, N); // Initialize h_C since beta=1.0

    std::cout << "Host matrice initialised" << std::endl;

    memcpy(h_C_cpu, h_C, size_C); // Copy initial h_C to h_C_cpu for CPU calculation


    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, size_A));
    CUDA_CHECK(hipMalloc(&d_B, size_B));
    CUDA_CHECK(hipMalloc(&d_C, size_C));
    std::cout << "Device memory allocated" << std::endl;

    CUDA_CHECK(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice)); // Copy initial h_C to device


    std::cout << "Input data copied from host to device" << std::endl;

    // Perform CPU calculation
    std::cout << "Performing CPU GEMM calculation..." << std::endl;
    gemm_cpu(h_A, h_B, h_C_cpu, M, N, K, alpha, beta);
    std::cout << "CPU GEMM calculation complete." << std::endl;


    // Cuda Event creation for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Defining the kernel dimensions

    dim3 blockSize(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridSize ((M+ BLOCK_SIZE - 1)/BLOCK_SIZE, (N + BLOCK_SIZE - 1)/BLOCK_SIZE);

    // Record start event
    CUDA_CHECK(hipEventRecord(start, 0));

    // Kernel Launch
    gemm_1d_blocktiling<BM, BN, BK, TM><<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    CUDA_CHECK(hipGetLastError());

    // Record stop event
    CUDA_CHECK(hipEventRecord(stop, 0));

    // Synchronize to ensure events are recorded
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Kernel execution time: " << milliseconds << "ms" << std::endl;

    // Destroying the events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));
    std::cout << "GPU Result copied back to host." << std::endl;

    // --- Verification ---
    std::cout << "Verifying GPU result against CPU result..." << std::endl;
    bool match = true;
    float tolerance = 1e-4; // Tolerance for floating-point comparison
    for (int i = 0; i < M * N; ++i) {
        if (fabs(h_C[i] - h_C_cpu[i]) > tolerance) {
            std::cerr << "Mismatch found at index " << i << ": GPU=" << h_C[i] << ", CPU=" << h_C_cpu[i] << std::endl;
            match = false;
            break; // Exit loop on first mismatch
        }
    }

    if (match) {
        std::cout << "Verification successful: GPU and CPU results match within tolerance." << std::endl;
    } else {
        std::cout << "Verification failed: GPU and CPU results do NOT match." << std::endl;
    }


    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cpu;

    std::cout<<"Host memory freed"<< std::endl;

    return 0;
}