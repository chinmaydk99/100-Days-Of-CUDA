#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>

#include <iostream>
#include <math.h>

#define BLOCK_DIM 128  // Must be <= embedding_dim

__global__ void LayerNorm_3D_Optimized(float* input, float* output, float* gamma, float* beta,
                                       int batch_size, int seq_length, int embedding_dim, float epsilon) {
    int batch_idx = blockIdx.x;  
    int seq_idx   = blockIdx.y;  
    int emb_idx   = threadIdx.x; 

    int index = (batch_idx * seq_length * embedding_dim) + (seq_idx * embedding_dim) + emb_idx;

    __shared__ float shared_mem[BLOCK_DIM];

    float val = input[index];

    shared_mem[emb_idx] = val;
    __syncthreads();

    for (int stride = BLOCK_DIM / 2; stride > 0; stride /= 2) {
        if (emb_idx < stride) {
            shared_mem[emb_idx] += shared_mem[emb_idx + stride];
        }
        __syncthreads();
    }

    float mean = 0.0f;
    if (emb_idx == 0) {
        mean = shared_mem[0] / embedding_dim;
    }
    __syncthreads();

    float diff = val - mean;
    shared_mem[emb_idx] = diff * diff;
    __syncthreads();

    for (int stride = BLOCK_DIM / 2; stride > 0; stride /= 2) {
        if (emb_idx < stride) {
            shared_mem[emb_idx] += shared_mem[emb_idx + stride];
        }
        __syncthreads();
    }

    float variance = 0.0f;
    if (emb_idx == 0) {
        variance = shared_mem[0] / embedding_dim;
    }
    __syncthreads();

    float stddev = sqrtf(variance + epsilon);
    output[index] = gamma[emb_idx] * ((val - mean) / stddev) + beta[emb_idx];
}

int main() {
    const int batch_size = 4, seq_length = 4, embedding_dim = 4;
    const int total_size = batch_size * seq_length * embedding_dim;

    float *Input_h, *Output_h, *Gamma_h, *Beta_h;

    Input_h = (float*)malloc(total_size * sizeof(float));
    Output_h = (float*)malloc(total_size * sizeof(float));
    Gamma_h = (float*)malloc(embedding_dim * sizeof(float));
    Beta_h = (float*)malloc(embedding_dim * sizeof(float));

    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < seq_length; j++) {
            for(int k = 0; k < embedding_dim; k++){
                Input_h[i * seq_length * embedding_dim + j * embedding_dim + k] = static_cast<float>(rand()) / RAND_MAX;
            }
        }
    }


    for(int i = 0; i < embedding_dim; i++){
        Gamma_h[i] = 1.0f;
        Beta_h[i] = 0.0f;
    }


    float *Input_d, *Output_d, *Gamma_d, *Beta_d;
    hipMalloc(&Input_d, total_size * sizeof(float));
    hipMalloc(&Output_d, total_size * sizeof(float));
    hipMalloc(&Gamma_d, embedding_dim * sizeof(float));
    hipMalloc(&Beta_d, embedding_dim * sizeof(float));


    hipMemcpy(Input_d, Input_h, total_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Gamma_d, Gamma_h, embedding_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Beta_d, Beta_h, embedding_dim * sizeof(float), hipMemcpyHostToDevice);

    hipMemset(Output_d, 0, total_size * sizeof(float));

    dim3 blocksize(embedding_dim);   
    dim3 gridsize(batch_size, seq_length);
    size_t shared_mem_size = emb_dim * sizeof(float); 

    LayerNorm_3D<<<gridsize, blocksize, shared_mem_size>>>(Input_d, Output_d, Gamma_d, Beta_d, batch_size, seq_length, embedding_dim, 1e-5);
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    hipMemcpy(Output_h, Output_d, total_size * sizeof(float), hipMemcpyDeviceToHost);

    printf("Layer Normalized Output:\n");
    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < seq_length; j++) {
            for(int k = 0; k < embedding_dim; k++){
                printf("%.2f ", Output_h[i * seq_length * embedding_dim + j * embedding_dim + k]);
            }
            printf("\n");
        }
        printf("\n");
    }

    // Free memory
    hipFree(Input_d);
    hipFree(Output_d);
    hipFree(Gamma_d);
    hipFree(Beta_d);
    free(Input_h);
    free(Output_h);
    free(Gamma_h);
    free(Beta_h);

    return 0;
}