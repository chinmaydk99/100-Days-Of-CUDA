#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  
#define THREADS_PER_BLOCK (N)

__global__ void SimpleSumReductionKernel(float *input, float *output){
    unsigned int i = threadIdx.x;

    for(int stride = blockDim.x / 2; stride >= 1; stride /= 2){
        if(threadIdx.x < stride){
            input[i] += input[i+stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0){
        *output = input[0];
    }
}

int main() {
    float h_input[N], h_output; 
    float *d_input, *d_output;  
    
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f; 
    }

    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    SimpleSumReductionKernel<<<1, THREADS_PER_BLOCK>>>(d_input, d_output);

    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);


    printf("Sum of array elements: %f\n", h_output);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}