#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  
#define THREADS_PER_BLOCK (N)

__global__ void SimpleSumReductionKernel(float *input, float *output){
    unsigned int i = threadIdx.x;

    for(int stride = 1; stride < blockDim.x; stride *= 2){
        if(threadIdx.x + stride < blockDim.x){
            input[i] += input[i+stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0){
        *output = input[0];
    }
}

int main() {
    float h_input[N], h_output;  // Host input array and result
    float *d_input, *d_output;   // Device input array and result
    
    // Initialize host input with values (for example, 1.0 for all elements)
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;  // Example: sum of N ones should be N
    }

    // Allocate memory on GPU
    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with one block of THREADS_PER_BLOCK threads
    SimpleSumReductionKernel<<<1, THREADS_PER_BLOCK>>>(d_input, d_output);

    // Copy result from device to host
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Sum of array elements: %f\n", h_output);

    // Free memory on GPU
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}