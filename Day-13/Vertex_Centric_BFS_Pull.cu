#include <iostream>
#include <hip/hip_runtime.h>
#include <climits>

struct CSCGraph{
    int numVertices;
    int numEdges;
    int *dstPtrs;
    int *src;
};

__global__ void bfs_pull(CSCGraph cscGraph, unsigned int *level, unsigned int *newVertexVisited, unsigned int currLevel){
    unsigned int vertex = blockDim.x*blockIdx.x + threadIdx.x;

    if(vertex < cscGraph.numVertices){
        if(level[vertex] == UINT_MAX){// If current node is univisited
            for(unsigned edge = cscGraph.dstPtrs[vertex]; edge < cscGraph.dstPtrs[vertex+1]; edge ++){
                unsigned int neighbor = cscGraph.src[edge];
                if(level[neighbor] == currLevel-1){
                    level[vertex] = currLevel;
                    *newVertexVisited = 1;
                    break;
                }
            }
        }
    }
}


int main() {
    const int numVertices = 4;
    const int numEdges = 4;

    int h_dstPtrs[] = {0, 1, 2, 3, 4};
    int h_src[] = {0, 0, 1, 2};

    unsigned int h_level[numVertices];
    std::fill_n(h_level, numVertices, UINT_MAX);
    h_level[0] = 0;

    CSCGraph h_cscGraph;
    h_cscGraph.numEdges = numEdges;
    h_cscGraph.numVertices = numVertices;

    hipMalloc(&h_cscGraph.src, numEdges*sizeof(int));
    hipMalloc(&h_cscGraph.dstPtrs, (numVertices+1)*sizeof(int));

    hipMemcpy(h_cscGraph.dstPtrs, h_dstPtrs, (numVertices + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(h_cscGraph.src, h_src, numEdges * sizeof(int), hipMemcpyHostToDevice);

    CSCGraph *d_cscGraph;
    hipMalloc(&d_cscGraph, sizeof(CSCGraph));
    hipMemcpy(d_cscGraph, &h_cscGraph, sizeof(CSCGraph), hipMemcpyHostToDevice);

    unsigned int *d_level, *d_newVertexVisited;
    hipMalloc(&d_level, numVertices * sizeof(unsigned int));
    hipMemcpy(d_level, h_level, numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc(&d_newVertexVisited, sizeof(unsigned int));

    unsigned int currLevel = 1;
    unsigned int newVertexVisited = 0;

    do {
        newVertexVisited = 0;
        hipMemcpy(d_newVertexVisited, &newVertexVisited, sizeof(unsigned int), hipMemcpyHostToDevice);

        bfs_pull<<<(numVertices + 255) / 256, 256>>>(*d_cscGraph, d_level, d_newVertexVisited, currLevel);
        hipMemcpy(&newVertexVisited, d_newVertexVisited, sizeof(unsigned int), hipMemcpyDeviceToHost);
        currLevel++;

    } while (newVertexVisited != 0);

    hipMemcpy(h_level, d_level, numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);

    std::cout << "Vertex Levels (CSC):\n";
    for (int i = 0; i < numVertices; i++) {
        std::cout << "Vertex " << i << ": Level " << h_level[i] << std::endl;
    }

    hipFree(d_cscGraph->dstPtrs);
    hipFree(d_cscGraph->src);
    hipFree(d_level);
    hipFree(d_newVertexVisited);

    return 0;
}

