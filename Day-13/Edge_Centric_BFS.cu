#include <iostream>
#include <hip/hip_runtime.h>
#include <climits>

struct COOGraph {
    int numVertices;
    int numEdges;
    int *dst;
    int *src;
};

__global__ void edge_bfs(COOGraph *coograph, unsigned int *level, unsigned int currLevel, unsigned int *newVertexVisited) {
    unsigned int edge = blockDim.x * blockIdx.x + threadIdx.x;

    if (edge < coograph->numEdges) {
        unsigned int vertex = coograph->src[edge];
        if (level[vertex] == currLevel - 1) {
            unsigned int neighbor = coograph->dst[edge];  // Fixed neighbor access
            if (level[neighbor] == UINT_MAX) {
                level[neighbor] = currLevel;
                *newVertexVisited = 1;
            }
        }
    }
}

int main() {
    const int numVertices = 4;
    const int numEdges = 4;

    int h_src[] = {0, 0, 1, 2};
    int h_dst[] = {1, 2, 3, 0};

    unsigned int h_level[numVertices];
    std::fill_n(h_level, numVertices, UINT_MAX);
    h_level[0] = 0;

    COOGraph h_cooGraph;
    h_cooGraph.numVertices = numVertices;
    h_cooGraph.numEdges = numEdges;

    hipMalloc(&h_cooGraph.src, numEdges * sizeof(int));
    hipMalloc(&h_cooGraph.dst, numEdges * sizeof(int));
    hipMemcpy(h_cooGraph.src, h_src, numEdges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(h_cooGraph.dst, h_dst, numEdges * sizeof(int), hipMemcpyHostToDevice);

    COOGraph *d_cooGraph;
    hipMalloc(&d_cooGraph, sizeof(COOGraph));
    hipMemcpy(d_cooGraph, &h_cooGraph, sizeof(COOGraph), hipMemcpyHostToDevice);

    unsigned int *d_level, *d_newVertexVisited;
    hipMalloc(&d_level, numVertices * sizeof(unsigned int));
    hipMemcpy(d_level, h_level, numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc(&d_newVertexVisited, sizeof(unsigned int));

    unsigned int currLevel = 1;
    unsigned int newVertexVisited = 0;

    do {
        newVertexVisited = 0;
        hipMemcpy(d_newVertexVisited, &newVertexVisited, sizeof(unsigned int), hipMemcpyHostToDevice);

        edge_bfs<<<(numEdges + 255) / 256, 256>>>(d_cooGraph, d_level, currLevel, d_newVertexVisited);

        hipMemcpy(&newVertexVisited, d_newVertexVisited, sizeof(unsigned int), hipMemcpyDeviceToHost);
        currLevel++;

    } while (newVertexVisited != 0);

    hipMemcpy(h_level, d_level, numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);

    std::cout << "Vertex Levels (COO):\n";
    for (int i = 0; i < numVertices; i++) {
        std::cout << "Vertex " << i << ": Level " << h_level[i] << std::endl;
    }

    // Free memory
    hipFree(h_cooGraph.src);
    hipFree(h_cooGraph.dst);
    hipFree(d_cooGraph);
    hipFree(d_level);
    hipFree(d_newVertexVisited);

    return 0;
}
