#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

using ElementAccumulator = int32_t;                
using ElementComputeEpilogue = float;             
using ElementInputA = cutlass::int4b_t;             
using ElementInputB = cutlass::int4b_t;             
using ElementOutput = cutlass::int4b_t;         
using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

using MMAOp = cutlass::arch::OpClassTensorOp;

using SmArch = cutlass::arch::Sm75;

using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 128>;  
using WarpShape = cutlass::gemm::GemmShape<64, 64, 128>;         
using InstructionShape = cutlass::gemm::GemmShape<8, 8, 32>;    
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

constexpr int NumStages = 2;

using EpilogueOp = cutlass::epilogue::thread::LinearCombinationClamp<
    ElementOutput,                                     
    8,                                                 
    ElementAccumulator,                                
    ElementComputeEpilogue>;                           

using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAddSaturate,
  cutlass::conv::IteratorAlgorithm::kAnalytic
>::Kernel;

using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    reference_check(false),
    measure_performance(true),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  bool valid() {
    int const kAlignment = 32;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {
      return false;
    }

    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {
      return false;
    }

    return true;
  }

  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c(); 

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }
  
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  double gflops(double runtime_s) const {
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result(): 
    runtime_ms(0), 
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out 
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

Result profile_convolution(Options const &options) {

  Result result;

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.input_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_c(options.output_size());

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8),
      0);

  cutlass::reference::host::TensorFill(
      tensor_c.host_view());

  cutlass::reference::host::TensorFill(
      tensor_ref_c.host_view());

  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_ref_c.sync_device();

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;
  int split_k_slices = 1;

  cutlass::conv::Conv2dProblemSize problem_size(      
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices);

  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_c.device_ref(),
    tensor_c.device_ref(),
    {options.alpha, options.beta},
  };

  ImplicitGemm implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

  if (options.reference_check) {
    std::cout << "Verification on host...\n";

    cutlass::reference::host::Conv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementComputeEpilogue,
      ElementAccumulator,
      ElementOutput,
      cutlass::NumericConverterClamp<ElementOutput, ElementComputeEpilogue>
    >(
      problem_size,
      tensor_a.host_ref(),
      tensor_b.host_ref(),
      tensor_c.host_ref(),
      tensor_ref_c.host_ref(),
      options.alpha,
      options.beta
    );

    tensor_c.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(
      tensor_c.host_view(),
      tensor_ref_c.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    }
    else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  }
  else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "09_tensor_conv_workspace_conv2dfprop_"
      << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c() 
      << "_"
      << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c() 
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace 
      << "Input = \n" << tensor_a.host_view() << "\n\n"
      << "Filters = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_c.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_c.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }

  if (options.measure_performance) {

    hipEvent_t events[2];
    
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_op();
      CUTLASS_CHECK(result.status);
    }

    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

int main(int argc, char const **args) {

  if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
    std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;
    return 0;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 7 || (props.major == 7 && props.minor >= 5))) {
    std::cerr << "Turing Tensor Ops must be run on a machine with compute capability at least 75."
              << std::endl;
    return 0;
  }

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {

    int batch_sizes[] = {1, 32, 64, 128, 256, 512};

    struct Benchmark {
      int h, w, c, k, r, s;
    } layers[] = {
      {56,  56,   64,   256, 1, 1},
      {56,  56,   64,    64, 1, 1},
      {56,  56,   64,    64, 3, 3},
      {56,  56,  256,    64, 1, 1},
      {56,  56,  256,   512, 1, 1},
      {56,  56,  256,   128, 1, 1},
      {28,  28,  128,   128, 3, 3},
      {28,  28,  128,   512, 1, 1},
      {28,  28,  512,   128, 1, 1},
      {28,  28,  512,  1024, 1, 1},
      {28,  28,  512,   256, 1, 1},
      {14,  14,  256,   256, 3, 3},
      {14,  14,  256,  1024, 1, 1},
      {14,  14,  1024,  256, 1, 1},
      {14,  14,  1024, 2048, 1, 1},
      {14,  14,  1024,  512, 1, 1},
      {7,    7,   512,  512, 3, 3},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {

        options.update({N, layer.h, layer.w, layer.c}, {layer.k, layer.r, layer.s, layer.c});

        Result result = profile_convolution(options);
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }

    Result result = profile_convolution(options);

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }
  return 0;
}