#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(){
     // Creating the cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int n =  4;
    double h_x[] = {1.0, 2.0, 3.0, 4.0};
    double h_y[] = {5.0, 6.0, 7.0, 8.0};

    printf("Inputs: ");
    printf("x:");
    for (int i = 0; i < n; i++) printf("%.1f ", h_x[i]); 
    printf("\n");

    printf("y:");
    for (int i = 0; i < n; i++) printf("%.1f ", h_y[i]); 
    printf("\n");

    double alpha = 1.0;

    double *d_x, *d_y;
    hipMalloc(&d_x, n*sizeof(double));
    hipMalloc(&d_y, n*sizeof(double));
    
    hipMemcpy(d_x, h_x, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n*sizeof(double), hipMemcpyHostToDevice);

    // D here is for double precision
    // This is doing alpha *x + y and result is stored in y
    hipblasDaxpy(handle, n, &alpha, d_x, 1, d_y, 1); // Here the 1 is stride between elements

    hipMemcpy(h_y, d_y, n*sizeof(double), hipMemcpyDeviceToHost);

    printf("y: ");
    for (int i = 0; i < n; i++) printf("%.1f ", h_y[i]); // [6.0, 8.0, 10.0, 12.0]
    printf("\n");

    hipFree(d_x); 
    hipFree(d_y);
    hipblasDestroy(handle);

    return 0;
}
   
