#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "book.h"

#define DIM 1024
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f
#define SPHERES 20

static void HandleError( hipError_t err,
    const char *file,
    int line ) {
if (err != hipSuccess) {
printf( "%s in %s at line %d\n", hipGetErrorString( err ),
file, line );
exit( EXIT_FAILURE );
}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct Sphere {
    float r, g, b;
    float radius;
    float x, y, z;

    __device__ float hit(float ox, float oy, float *n) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / radius;
            return dz + z;
        }
        return -INF;
    }
};

__constant__ Sphere s[SPHERES];

__global__ void kernel(unsigned char *ptr) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float ox = x - DIM / 2;
    float oy = y - DIM / 2;

    float r = 0, g = 0, b = 0;
    float maxz = -INF;

    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void save_ppm(const char *filename, unsigned char *data, int width, int height) {
    std::ofstream f(filename, std::ios::binary);
    f << "P6\n" << width << " " << height << "\n255\n";
    for (int i = 0; i < width * height; ++i) {
        f.put(data[i * 4 + 0]);  // R
        f.put(data[i * 4 + 1]);  // G
        f.put(data[i * 4 + 2]);  // B
    }
    f.close();
}

int main() {
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    unsigned char *dev_bitmap;
    unsigned char *host_bitmap = new unsigned char[DIM * DIM * 4];

    // Allocate device memory for pixel buffer
    HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, DIM * DIM * 4));

    // Generate sphere data
    Sphere *temp_s = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(1000.0f) - 500;
        temp_s[i].y = rnd(1000.0f) - 500;
        temp_s[i].z = rnd(1000.0f) - 500;
        temp_s[i].radius = rnd(100.0f) + 20;
    }

    // Copy to constant memory
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere) * SPHERES));
    free(temp_s);

    // Launch kernel
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>(dev_bitmap);

    // Copy result back to host
    HANDLE_ERROR(hipMemcpy(host_bitmap, dev_bitmap, DIM * DIM * 4, hipMemcpyDeviceToHost));

    // Save as PPM file
    save_ppm("output.ppm", host_bitmap, DIM, DIM);
    std::cout << "Saved output.ppm\n";

    // Print timing info
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));
    std::cout << "Render time: " << elapsed << " ms\n";

    // Cleanup
    HANDLE_ERROR(hipFree(dev_bitmap));
    delete[] host_bitmap;
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return 0;
}
